#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include <cassert>
#include <cstdio>
#include <cmath>
#include <GL/glut.h>

static int counter = 0;

template <int X, int Y, class T = float>
struct volume {
    T *grid;

    void allocate() {
        size_t size = X * Y;
        checkCudaErrors(hipMallocManaged(&grid, size * sizeof(T)));
    }

    void free() {
        checkCudaErrors(hipFree(grid));
    }

    __host__ __device__ T &at(int i, int j) const {
        return grid[i + j * X];
    }

    __host__ __device__ auto &at(int c, int i, int j) const {
        return at(i, j)[c];
    }
};

#define GSL(_, start, end) \
    int _ = (start) + blockDim._ * blockIdx._ + threadIdx._; \
    _ < (end); _ += blockDim._ * gridDim._


static inline const float ka = 2.0f;
static inline const float ga = 0.2f;

template <int X, int Y>
struct DOM {
    volume<X, Y, float> pos;
    volume<X, Y, float> vel;
    volume<X, Y, uint8_t> mask;
    volume<X, Y, uint8_t> active;

    void allocate() {
        pos.allocate();
        vel.allocate();
        mask.allocate();
        active.allocate();
    }

    __device__ float laplacian(int i, int j) const {
        auto const dx = 10.f / X;
        return (-4 * pos.at(i, j) + pos.at(i, j - 1) + pos.at(i, j + 1)
            + pos.at(i + 1, j) + pos.at(i - 1, j)) / (4 * dx * dx);
    }
};

template <int X, int Y>
__global__ void initialize1(DOM<X, Y> dom, int type) {
    for (GSL(y, 0, Y)) for (GSL(x, 0, X)) {
        dom.vel.at(x, y) = 0.f;
        dom.pos.at(x, y) = 0.f;
        dom.mask.at(x, y) = 0;
        float fx = x * 2.f / X - 1.f;
        float fy = y * 2.f / Y - 1.f;
        float f2 = fx * fx + fy * fy;
        if (f2 < 0.1f || x == 0 || x == X - 1 || y == 0 || y == Y - 1) {
            dom.mask.at(x, y) = 1;
        }
        if (type == 0) {  // hi grid
            dom.active.at(x, y) = (fx < +0.05f);
        } else {  // lo grid
            dom.active.at(x, y) = (fx > -0.05f);
        }
    }
}

template <int X, int Y>
void initialize(DOM<X, Y> dom, int type) {
    initialize1<<<dim3(X / 16, Y / 16, 1), dim3(16, 16, 1)>>>(dom, type);
}

template <int X, int Y>
__global__ void substep1(DOM<X, Y> dom) {
    for (GSL(y, 0, Y)) for (GSL(x, 0, X)) {
        auto const dt = 5.f / X;
        if (!dom.active.at(x, y)) continue;
        if (dom.mask.at(x, y) != 0)
            continue;
        float acc = ka * dom.laplacian(x, y) - ga * dom.vel.at(x, y);
        dom.vel.at(x, y) += acc * dt;
    }
}

template <int X, int Y>
__global__ void substep2(DOM<X, Y> dom) {
    for (GSL(y, 0, Y)) for (GSL(x, 0, X)) {
        auto const dt = 5.f / X;
        if (!dom.active.at(x, y)) continue;
        dom.pos.at(x, y) += dom.vel.at(x, y) * dt;
    }
}

template <int X, int Y>
__global__ void substep3(DOM<X, Y> dom, float height) {
    for (GSL(y, 0, Y)) for (GSL(x, 0, X)) {
        if (!dom.active.at(x, y)) continue;
        float fx = x * 2.f / X - .25f;
        float fy = y * 2.f / Y - .25f;
        float f2 = fx * fx + fy * fy;
        if (f2 < 0.01f) {
            dom.pos.at(x, y) = height;
        }
    }
}

template <int X, int Y>
void substep(DOM<X, Y> dom) {
    substep1<<<dim3(X / 16, Y / 16, 1), dim3(16, 16, 1)>>>(dom);
    substep2<<<dim3(X / 16, Y / 16, 1), dim3(16, 16, 1)>>>(dom);
    float height = 1.0f * sinf(counter * 0.08f);
    substep3<<<dim3(X / 16, Y / 16, 1), dim3(16, 16, 1)>>>(dom, height);
}

template <int X, int Y>
__global__ void upper1(volume<X * 2, Y * 2> hi, volume<X, Y> lo,
    volume<X * 2, Y * 2, uint8_t> hi_active, volume<X, Y, uint8_t> lo_active) {
    for (GSL(y, 0, Y)) for (GSL(x, 0, X)) {
        if (!lo_active.at(x, y) || !hi_active.at(x * 2, y * 2)) continue;
        float val = lo.at(x, y);
        for (int dy = 0; dy < 2; dy++) for (int dx = 0; dx < 2; dx++) {
            hi.at(x * 2 + dx, y * 2 + dy) = val;
        }
    }
}

template <int X, int Y>
void upper(volume<X * 2, Y * 2> hi, volume<X, Y> lo,
    volume<X * 2, Y * 2, uint8_t> hi_active, volume<X, Y, uint8_t> lo_active) {
    upper1<<<dim3(X / 16, Y / 16, 1), dim3(16, 16, 1)>>>(hi, lo,
        hi_active, lo_active);
}

template <int X, int Y>
__global__ void lower1(volume<X * 2, Y * 2> hi, volume<X, Y> lo,
    volume<X * 2, Y * 2, uint8_t> hi_active, volume<X, Y, uint8_t> lo_active) {
    for (GSL(y, 0, Y)) for (GSL(x, 0, X)) {
        if (!lo_active.at(x, y) || !hi_active.at(x * 2, y * 2)) continue;
        float val = 0.f;
        for (int dy = 0; dy < 2; dy++) for (int dx = 0; dx < 2; dx++) {
            val += hi.at(x * 2 + dx, y * 2 + dy);
        }
        lo.at(x, y) = val * 0.25f;
    }
}

template <int X, int Y>
void lower(volume<X * 2, Y * 2> hi, volume<X, Y> lo,
    volume<X * 2, Y * 2, uint8_t> hi_active, volume<X, Y, uint8_t> lo_active) {
    lower1<<<dim3(X / 16, Y / 16, 1), dim3(16, 16, 1)>>>(hi, lo,
        hi_active, lo_active);
}

template <int X, int Y>
void lower(DOM<X * 2, Y * 2> hi, DOM<X, Y> lo) {
    lower(hi.pos, lo.pos, hi.active, lo.active);
    lower(hi.vel, lo.vel, hi.active, lo.active);
}

template <int X, int Y>
void upper(DOM<X * 2, Y * 2> hi, DOM<X, Y> lo) {
    upper(hi.pos, lo.pos, hi.active, lo.active);
    upper(hi.vel, lo.vel, hi.active, lo.active);
}

#define NX 512
#define NY 512
DOM<NX / 1, NY / 1> dom;
DOM<NX / 2, NY / 2> dom2;
float *pixels;

void initFunc() {
    checkCudaErrors(hipMallocManaged(&pixels, NX * NY * sizeof(float)));
    dom.allocate();
    dom2.allocate();
    initialize(dom, 0);
    initialize(dom2, 1);
}

void stepFunc() {
    substep(dom);
    substep(dom);
    lower(dom, dom2);
    substep(dom2);
    upper(dom, dom2);
    counter++;
}

template <int X, int Y>
__global__ void render1(float *pixels, DOM<X, Y> dom, float scale) {
    for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        float val = dom.pos.at(x * X / NX, y * Y / NY) * scale;
        pixels[y * NX + x] = 0.5f + 0.5f * val;
    }
}

template <int X, int Y>
void render(float *pixels, DOM<X, Y> dom, float scale) {
    render1<<<dim3(NX / 16, NY / 16, 1), dim3(16, 16, 1)>>>(pixels, dom, scale);
}

void renderFunc() {
    if (counter % 200 < 100) {
        render(pixels, dom, 1.f);
    } else {
        render(pixels, dom2, 1.f);
    }
    checkCudaErrors(hipDeviceSynchronize());
    /*printf("03:%f\n", pixels[0 * N + 3]);
    printf("30:%f\n", pixels[3 * NX + 0]);
    printf("33:%f\n", pixels[3 * NX + 3]);*/
}

void displayFunc() {
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(NX, NY, GL_RED, GL_FLOAT, pixels);
    glFlush();
}

#define ITV 2
void timerFunc(int unused) {
    stepFunc();
    renderFunc();
    glutPostRedisplay();
    glutTimerFunc(ITV, timerFunc, 0);
}

void keyboardFunc(unsigned char key, int x, int y) {
    if (key == 27)
        exit(0);
}

int main(int argc, char **argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DEPTH | GLUT_SINGLE | GLUT_RGBA);
    glutInitWindowPosition(100, 100);
    glutInitWindowSize(NX, NY);
    glutCreateWindow("GLUT Window");
    glutDisplayFunc(displayFunc);
    glutKeyboardFunc(keyboardFunc);
    initFunc();
    renderFunc();
    glutTimerFunc(ITV, timerFunc, 0);
    glutMainLoop();
}
