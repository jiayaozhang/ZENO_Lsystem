#include "hip/hip_runtime.h"
#if 0
#include <stdio.h>
__global__ void test() { printf("FuCK U NVIDIA!\n"); } int main(void) { test<<<1, 1>>>(); hipDeviceSynchronize(); }
#else

#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include <cassert>
#include <cstdio>
#include <cmath>


int main(void)
{
  int nx = 32;
  float *img = nullptr;
  checkCudaErrors(hipMallocManaged(&img, nx * sizeof(float)));

  for (int i = 0; i < nx; i++) {
    img[i] = drand48();
  }

  checkCudaErrors(hipDeviceSynchronize());

  for (int i = 0; i < nx; i++) {
    printf("%f\n", img[i]);
  }

  return 0;
}
#endif
